#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */




#include <poggers/allocators/templated_bitbuddy.cuh>

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>


#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace poggers::allocators;



template <typename allocator>
__global__ void test_single_thread_malloc_only(allocator * alloc, uint64_t num_allocs){


   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid != 0) return;

   for (uint64_t i = 0; i < num_allocs; i++){

      uint64_t test_val = alloc->malloc_offset(1);

      if (test_val == (~0ULL)){printf("malloc Error\n"); }

      else {


      if (!alloc->free(test_val)) printf("Free Error\n");


      }
      //printf("i/offset: %llu / %llu\n", i, test_val);

   }

}


template <typename allocator>
__global__ void test_multi_thread_malloc_only(allocator * alloc, uint64_t num_allocs){


   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_allocs) return;

   uint64_t test_val = alloc->malloc_offset(1);


   if (test_val != (~0ULL)){

      alloc->free(test_val);
   } else { 

      printf("Fail!\n");

   }

   //printf("i/offset: %llu / %llu\n", tid, test_val);

}


template <typename allocator>
__global__ void test_multi_thread_rounds(allocator * alloc, uint64_t num_allocs, uint64_t num_rounds){


   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_allocs) return;


   for (uint64_t i =0; i < num_rounds; i++){


   uint64_t test_val = alloc->malloc_offset(1);


   if (test_val != (~0ULL)){

      alloc->free(test_val);

   } else { 

      printf("Fail!\n");
   }


   }

   //printf("Finished with %llu\n", tid);

   //printf("i/offset: %llu / %llu\n", tid, test_val);

}


template <typename allocator>
__host__ void test_multi_thread_alloc(uint64_t num_allocs){



   allocator * alloc = allocator::generate_on_device();


   hipDeviceSynchronize();

   test_multi_thread_malloc_only<allocator><<<(num_allocs-1)/1024+1,1024>>>(alloc, num_allocs);

   hipDeviceSynchronize();

   allocator::free_on_device(alloc);


   printf("Done with multi %llu\n", num_allocs);
}


template <typename allocator>
__host__ void test_multi_thread_alloc_rounds(uint64_t num_allocs, uint64_t num_rounds){



   allocator * alloc = allocator::generate_on_device();


   hipDeviceSynchronize();

   auto rounds_start = std::chrono::high_resolution_clock::now();

   test_multi_thread_rounds<allocator><<<(num_allocs-1)/1024+1,1024>>>(alloc, num_allocs, num_rounds);

   hipDeviceSynchronize();

   auto rounds_end = std::chrono::high_resolution_clock::now();




   std::chrono::duration<double> rounds_diff = rounds_end-rounds_start;

   allocator::free_on_device(alloc);

   printf("Done with multi rounds %llu %llu\n", num_allocs, num_rounds);

   std::cout << "Inserted " << num_allocs*num_rounds << " in " << rounds_diff.count() << " seconds\n";

   printf("Malloc/Free pair throughput: %f \n", 1.0*num_allocs*num_rounds/rounds_diff.count());
 

}



template <typename allocator>
__host__ void test_single_thread_alloc(uint64_t num_allocs){



   allocator * alloc = allocator::generate_on_device();


   hipDeviceSynchronize();

   test_single_thread_malloc_only<allocator><<<1,1>>>(alloc, num_allocs);

   hipDeviceSynchronize();

   allocator::free_on_device(alloc);
}


//using allocator_type = buddy_allocator<0,0>;

int main(int argc, char** argv) {


   using allocator = templated_bitbuddy<0,32>;

   test_single_thread_alloc<allocator>(1);

   test_single_thread_alloc<allocator>(32);

   using allocator_1 = templated_bitbuddy<1,1024>;


   //test_multi_thread_alloc<allocator_1>(1024);

   test_multi_thread_alloc_rounds<allocator_1>(1024, 10);


   using allocator_2 = templated_bitbuddy<2,32768>;

   //test_multi_thread_alloc<allocator_2>(32768);

   test_multi_thread_alloc_rounds<allocator_2>(32768, 10);


   using allocator_3 = templated_bitbuddy<3, 1048576>;

   test_multi_thread_alloc_rounds<allocator_3>(1048576, 10);

  


   return 0;

}
