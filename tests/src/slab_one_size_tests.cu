#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */




#include <poggers/allocators/slab_one_size.cuh>

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>

using namespace std::chrono;


#include <hip/hip_cooperative_groups.h>


#include <poggers/allocators/one_size_allocator.cuh>

namespace cg = cooperative_groups;

using namespace poggers::allocators;


double elapsed(high_resolution_clock::time_point t1, high_resolution_clock::time_point t2) {
   return (duration_cast<duration<double> >(t2 - t1)).count();
}



__global__ void malloc_tests(one_size_slab_allocator<15> * allocator, uint64_t max_mallocs){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= max_mallocs) return;

   void * allocation = allocator->malloc();


   return;

}


__host__ void boot_slab_one_size(){


   one_size_slab_allocator<15> * test_alloc = one_size_slab_allocator<15>::generate_on_device(64000000, 16);

   hipDeviceSynchronize();

   malloc_tests<<<1, 256>>>(test_alloc, 10);

   hipDeviceSynchronize();


   one_size_slab_allocator<15>::free_on_device(test_alloc);

   hipDeviceSynchronize();

}

template <int num_blocks>
__global__ void allocate_into_array(one_size_slab_allocator<num_blocks> * allocator, uint64_t * array, uint64_t num_mallocs, uint64_t * misses){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_mallocs) return;

   void * allocation = allocator->malloc();

   if (allocation != nullptr){

      uint64_t offset = allocator->get_offset_from_ptr(allocation);

      if (offset >= allocator->get_largest_allocation_offset()){

         printf("allocation bug %llx > %llx\n", offset, num_mallocs+15000000);
      }

      char * cast = (char *) allocation;

      cast[0] = 't';
   
   } else {
      atomicAdd((unsigned long long int *) misses, 1ULL);
   }

   array[tid] = (uint64_t) allocation;

   //printf("Tid %llu\n", tid);


}


template <int num_blocks>
__global__ void allocate_into_array_bits(one_size_slab_allocator<num_blocks> * allocator, uint64_t * array, uint64_t num_mallocs, uint64_t * misses){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_mallocs) return;

   void * allocation = allocator->malloc_mark_unpinned(&misses[3]);

   if (allocation != nullptr){

      uint64_t offset = allocator->get_offset_from_ptr(allocation);

      if (offset >= allocator->get_largest_allocation_offset()){

         printf("allocation bug %llu > %llu, diff is %llu\n", offset, num_mallocs+15000000, offset - (num_mallocs+15000000));
      }

      char * cast = (char *) allocation;

      cast[0] = 't';


      int local_bits = offset % 64;
      uint64_t shrunken_offset = offset/64;

      uint64_t old_bits = atomicOr((unsigned long long int *)&array[shrunken_offset], (1ULL << local_bits));

      if (old_bits & (1ULL << local_bits)){
         //printf("Bit alloc array Bug with index %llu\n", offset);
         atomicAdd((unsigned long long int *)misses, 1ULL);
      }

   
   } else {
      atomicAdd((unsigned long long int *) misses, 1ULL);
   }

   //printf("Tid %llu\n", tid);


}


template <int num_blocks>
__global__ void free_from_array(one_size_slab_allocator<num_blocks> * allocator, uint64_t * array, uint64_t num_mallocs){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_mallocs) return;

   void * allocation = (void *) array[tid];

   if (allocation != nullptr){
      //allocator->free(allocation, num_mallocs+15000000);
      allocator->free(allocation);
   }



}

template <int num_blocks>
__global__ void free_from_array_bits(one_size_slab_allocator<num_blocks> * allocator, uint64_t * array, uint64_t max_offset_bit, uint64_t * misses){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= max_offset_bit) return;


   uint64_t lower_offset = tid/64;
   int offset_bit = tid % 64;

   bool valid = (array[lower_offset] & (1ULL << offset_bit));

   if (!valid) return;

   void * allocation = (void *) (allocator->offset_size*tid + allocator->extra_memory);

   if (allocator->get_offset_from_ptr(allocation) != tid){
      printf("Bug in free offset generation\n");
   }

   allocator->free_count_misses(allocation, misses);



}

template <int num_blocks>
__global__ void log_free_kernel(one_size_slab_allocator<num_blocks> * allocator, uint64_t * array, uint64_t max_offset_bit){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= max_offset_bit) return;

   uint64_t lower_offset = tid/64;
   int offset_bit = tid % 64;

   bool valid = (array[lower_offset] & (1ULL << offset_bit));

   if (!valid) return;

   void * allocation = (void *) (allocator->offset_size*tid + allocator->extra_memory);


   allocator->log_free();


}


template <int blocks>
__host__ void test_num_malloc_frees_bitarr(uint64_t num_mallocs, int num_rounds){

      //I think 4,000,000 is enough to saturate the wavefront.
   //108 SMs, each with 4096 items
   //pocket math says 500,000 is sufficient for an A100
   //times 16 jk gonna do 10,000,000 to be safe

   printf("Starting test with %llu threads and %d rounds\n", num_mallocs, num_rounds);


   uint64_t total_num_allocs = 15000000+num_mallocs+4096;

   uint64_t total_num_allocs_bits = (total_num_allocs-1)/64+1;


   one_size_slab_allocator<blocks> * test_alloc = one_size_slab_allocator<blocks>::generate_on_device(total_num_allocs, 1);


   uint64_t pre_fill = test_alloc->report_fill();

   uint64_t pre_max = test_alloc->report_max();

   printf("Initial fill ratio %llu/%llu %f \n", pre_fill, pre_max, 1.0*pre_fill/pre_max);


   uint64_t * array;

   hipMalloc((void ** )&array, sizeof(uint64_t)*total_num_allocs_bits);

   hipMemset(array, 0, sizeof(uint64_t)*total_num_allocs_bits);

   uint64_t * misses;

   hipMallocManaged((void **)&misses, sizeof(uint64_t)*4);

   hipDeviceSynchronize();



   for (int i=0; i< num_rounds; i++){

      misses[0] = 0;
      misses[1] = 0;
      misses[2] = 0;
      misses[3] = 0;

      hipDeviceSynchronize();



      allocate_into_array_bits<blocks><<<(num_mallocs -1)/512+1, 512>>>(test_alloc, array, num_mallocs, misses);

      hipDeviceSynchronize();

      uint64_t half_fill = test_alloc->report_fill();

      uint64_t half_max = test_alloc->report_max();

      printf("Halfway through iteration %d: %llu/%llu %f \n", i, half_fill, half_max, 1.0*half_fill/half_max);



      //log_free_kernel<blocks><<<(num_mallocs-1)/512+1,512


      free_from_array_bits<blocks><<<(total_num_allocs -1)/512+1, 512>>>(test_alloc, array, total_num_allocs, misses+1);

      hipDeviceSynchronize();

      hipMemset(array, 0, sizeof(uint64_t)*total_num_allocs_bits);

      hipDeviceSynchronize();

      uint64_t fill = test_alloc->report_fill();
      uint64_t max = test_alloc->report_max();

      printf("Done with cycle %d, %llu/%llu: %f misses. %llu/%llu free\n", i, misses[0], num_mallocs, 1.0*misses[0]/num_mallocs, fill, max);
      printf("Misses in free: blocks unpinned %llu, blocks freed %llu, threads that failed %llu\n", misses[3], misses[1], misses[2]);


   }

   hipFree(array);

   one_size_slab_allocator<blocks>::free_on_device(test_alloc);

   return;



}


template <int num_blocks>
__host__ void test_num_malloc_frees(uint64_t num_mallocs, int num_rounds){

   //I think 4,000,000 is enough to saturate the wavefront.
   //108 SMs, each with 4096 items
   //pocket math says 500,000 is sufficient for an A100
   //times 16 jk gonna do 10,000,000 to be safe

   high_resolution_clock::time_point malloc_start, malloc_end, free_start, free_end;

   printf("Starting test with %llu threads and %d rounds\n", num_mallocs, num_rounds);

   one_size_slab_allocator<num_blocks> * test_alloc = one_size_slab_allocator<num_blocks>::generate_on_device(15000000+num_mallocs, 1);


   uint64_t pre_fill = test_alloc->report_fill();

   uint64_t pre_max = test_alloc->report_max();

   printf("Initial fill ratio %llu/%llu %f \n", pre_fill, pre_max, 1.0*pre_fill/pre_max);


   uint64_t * array;

   hipMalloc((void ** )&array, sizeof(uint64_t)*num_mallocs);

   uint64_t * misses;

   hipMallocManaged((void **)&misses, sizeof(uint64_t));

   hipDeviceSynchronize();



   for (int i=0; i< num_rounds; i++){

      misses[0] = 0;

      hipDeviceSynchronize();

      malloc_start = high_resolution_clock::now();

      allocate_into_array<num_blocks><<<(num_mallocs -1)/512+1, 512>>>(test_alloc, array, num_mallocs, misses);

      hipDeviceSynchronize();

      malloc_end = high_resolution_clock::now();

      uint64_t half_fill = test_alloc->report_fill();

      uint64_t half_max = test_alloc->report_max();

      printf("Halfway through iteration %d: %llu/%llu %f \n", i, half_fill, half_max, 1.0*half_fill/half_max);

      hipDeviceSynchronize();

      free_start = high_resolution_clock::now();

      free_from_array<num_blocks><<<(num_mallocs -1)/512+1, 512>>>(test_alloc, array, num_mallocs);

      hipDeviceSynchronize();

      free_end = high_resolution_clock::now();

      uint64_t fill = test_alloc->report_fill();
      uint64_t max = test_alloc->report_max();

      printf("Done with cycle %d. %llu/%llu: %f misses. %llu/%llu free\n", i, misses[0], num_mallocs, 1.0*misses[0]/num_mallocs, fill, max);
      std::cout << "Cycle took " << elapsed(malloc_start, malloc_end) << " for malloc and " << elapsed(free_start, free_end) << " for frees.\n";


   }

   hipFree(array);

   one_size_slab_allocator<num_blocks>::free_on_device(test_alloc);

   return;



}


__host__ void test_num_malloc_no_free(uint64_t num_mallocs, int num_rounds){

    printf("Starting test with %llu threads and %d rounds\n", num_mallocs, num_rounds);


   uint64_t * array;

   hipMalloc((void ** )&array, sizeof(uint64_t)*num_mallocs);

   uint64_t * misses;

   hipMallocManaged((void **)&misses, sizeof(uint64_t));

   hipDeviceSynchronize();


   for (int i=0; i< num_rounds; i++){


      misses[0] = 0;

      hipDeviceSynchronize();

      one_size_slab_allocator<15> * test_alloc = one_size_slab_allocator<15>::generate_on_device(15000000+num_mallocs, 1);

      hipDeviceSynchronize();

      allocate_into_array<15><<<(num_mallocs -1)/512+1, 512>>>(test_alloc, array, num_mallocs, misses);

      hipDeviceSynchronize();

      //free_from_array<<<(num_mallocs -1)/512+1, 512>>>(test_alloc, array, num_mallocs);

      one_size_slab_allocator<15>::free_on_device(test_alloc);

      hipDeviceSynchronize();

      printf("Done with cycle %d, misses %llu\n", i, misses[0]);


   }

   hipFree(array);



   return;


}



//using allocator_type = buddy_allocator<0,0>;

int main(int argc, char** argv) {


   // for (int i =0; i< 20; i++){
   //    boot_slab_one_size();
   // }
   
   //test_num_malloc_frees(1000, 10000);


   //test_num_malloc_frees(10000, 100);

   //test_num_malloc_frees(10000, 10);

   test_num_malloc_frees<4>(1000000000, 10);

   //test_num_malloc_frees(1000000, 10);

   //test_num_malloc_frees(100000000, 10);

   //test_num_malloc_frees_bitarr<4>(100000000, 10);



 
   hipDeviceReset();
   return 0;

}
